#include "hip/hip_runtime.h"

#include "headers/gpu-crack.h"

/* 
    Optimized SHA1 function written by author of Pyrit:
    (I added some comments to his code to explain the optimizations he did)
    
    Quote from his comments:
    "This is a 'special-version' of the SHA1 round function. *ctx is the current state,
     that gets updated by *data. Notice the lack of endianess-changes here.
     This SHA1-implementation follows the more-instructions-less-space paradigm, since registers
     and (fast) memory on the device are precious, threads are not. Only the starting values
     of W[0] to W[4] are defined by parameters. We fix the rest to invariant values and leave
     the possible register allocation optimization to the compiler."
*/
__device__
void sha1_process ( const SHA_DEV_CTX *ctx , SHA_DEV_CTX *data ) {

    uint32_t temp, W[16], A, B, C, D, E;

    // W[5] contains the appended '1' bit
    // W[15] is the total length of buffer being hashed: 64 byte keypad + 20 byte prior hash
    W[ 0] = data->h0; W[ 1] = data->h1;
    W[ 2] = data->h2; W[ 3] = data->h3;
    W[ 4] = data->h4; W[ 5] = 0x80000000;
    W[ 6] = 0; W[ 7] = 0;
    W[ 8] = 0; W[ 9] = 0;
    W[10] = 0; W[11] = 0;
    W[12] = 0; W[13] = 0;
    W[14] = 0; W[15] = (64+20)*8;

    // Values from the previous round
    A = ctx->h0;
    B = ctx->h1;
    C = ctx->h2;
    D = ctx->h3;
    E = ctx->h4;

// This is a 'rotateleft by n bits' macro
#undef S
#define S(x,n) ((x << n) | (x >> (32 - n)))

// In this special SHA1, we do not need to actually extend to 80 words, since
// each word ends up being constructed from the initial 16 words.
// (Try it out on paper and see yourself)
// The 0x0F translates the index into the first 16 words.
#undef R
#define R(t)                                            \
(                                                       \
    temp = W[(t -  3) & 0x0F] ^ W[(t - 8) & 0x0F] ^     \
           W[(t - 14) & 0x0F] ^ W[ t      & 0x0F],      \
    ( W[t & 0x0F] = S(temp,1) )                         \
)

// The calculation of 'temp' in wiki pseudocode
#undef P
#define P(a,b,c,d,e,x)                                  \
{                                                       \
    e += S(a,5) + F(b,c,d) + K + x; b = S(b,30);        \
}

// Note that the ABCDE rotation is done manually here (80 rotations total)
// This is much more efficient than using a for loop and doing rotations
// using assignments in each iteration
#define F(x,y,z) (z ^ (x & (y ^ z)))
#define K 0x5A827999

    P( A, B, C, D, E, W[0]  );
    P( E, A, B, C, D, W[1]  );
    P( D, E, A, B, C, W[2]  );
    P( C, D, E, A, B, W[3]  );
    P( B, C, D, E, A, W[4]  );
    P( A, B, C, D, E, W[5]  );
    P( E, A, B, C, D, W[6]  );
    P( D, E, A, B, C, W[7]  );
    P( C, D, E, A, B, W[8]  );
    P( B, C, D, E, A, W[9]  );
    P( A, B, C, D, E, W[10] );
    P( E, A, B, C, D, W[11] );
    P( D, E, A, B, C, W[12] );
    P( C, D, E, A, B, W[13] );
    P( B, C, D, E, A, W[14] );
    P( A, B, C, D, E, W[15] );
    P( E, A, B, C, D, R(16) );
    P( D, E, A, B, C, R(17) );
    P( C, D, E, A, B, R(18) );
    P( B, C, D, E, A, R(19) );

#undef K
#undef F

#define F(x,y,z) (x ^ y ^ z)
#define K 0x6ED9EBA1

    P( A, B, C, D, E, R(20) );
    P( E, A, B, C, D, R(21) );
    P( D, E, A, B, C, R(22) );
    P( C, D, E, A, B, R(23) );
    P( B, C, D, E, A, R(24) );
    P( A, B, C, D, E, R(25) );
    P( E, A, B, C, D, R(26) );
    P( D, E, A, B, C, R(27) );
    P( C, D, E, A, B, R(28) );
    P( B, C, D, E, A, R(29) );
    P( A, B, C, D, E, R(30) );
    P( E, A, B, C, D, R(31) );
    P( D, E, A, B, C, R(32) );
    P( C, D, E, A, B, R(33) );
    P( B, C, D, E, A, R(34) );
    P( A, B, C, D, E, R(35) );
    P( E, A, B, C, D, R(36) );
    P( D, E, A, B, C, R(37) );
    P( C, D, E, A, B, R(38) );
    P( B, C, D, E, A, R(39) );

#undef K
#undef F

#define F(x,y,z) ((x & y) | (z & (x | y)))
#define K 0x8F1BBCDC

    P( A, B, C, D, E, R(40) );
    P( E, A, B, C, D, R(41) );
    P( D, E, A, B, C, R(42) );
    P( C, D, E, A, B, R(43) );
    P( B, C, D, E, A, R(44) );
    P( A, B, C, D, E, R(45) );
    P( E, A, B, C, D, R(46) );
    P( D, E, A, B, C, R(47) );
    P( C, D, E, A, B, R(48) );
    P( B, C, D, E, A, R(49) );
    P( A, B, C, D, E, R(50) );
    P( E, A, B, C, D, R(51) );
    P( D, E, A, B, C, R(52) );
    P( C, D, E, A, B, R(53) );
    P( B, C, D, E, A, R(54) );
    P( A, B, C, D, E, R(55) );
    P( E, A, B, C, D, R(56) );
    P( D, E, A, B, C, R(57) );
    P( C, D, E, A, B, R(58) );
    P( B, C, D, E, A, R(59) );

#undef K
#undef F

#define F(x,y,z) (x ^ y ^ z)
#define K 0xCA62C1D6

    P( A, B, C, D, E, R(60) );
    P( E, A, B, C, D, R(61) );
    P( D, E, A, B, C, R(62) );
    P( C, D, E, A, B, R(63) );
    P( B, C, D, E, A, R(64) );
    P( A, B, C, D, E, R(65) );
    P( E, A, B, C, D, R(66) );
    P( D, E, A, B, C, R(67) );
    P( C, D, E, A, B, R(68) );
    P( B, C, D, E, A, R(69) );
    P( A, B, C, D, E, R(70) );
    P( E, A, B, C, D, R(71) );
    P( D, E, A, B, C, R(72) );
    P( C, D, E, A, B, R(73) );
    P( B, C, D, E, A, R(74) );
    P( A, B, C, D, E, R(75) );
    P( E, A, B, C, D, R(76) );
    P( D, E, A, B, C, R(77) );
    P( C, D, E, A, B, R(78) );
    P( B, C, D, E, A, R(79) );

#undef K
#undef F

    data->h0 = ctx->h0 + A;
    data->h1 = ctx->h1 + B;
    data->h2 = ctx->h2 + C;
    data->h3 = ctx->h3 + D;
    data->h4 = ctx->h4 + E;
}

// GPU Kernel called by the Host Cracking Thread
// This is similar to pyrit's code
__global__
void crack_gpu_kernel ( kernel_input_buffer *inbuffer , kernel_output_buffer *outbuffer , int max_num ) {

    // Loop variable
    int i;

    // Hash value of previous round
    SHA_DEV_CTX prev_ctx;
    // Part of the PMK (20 bytes first part, 12 bytes second part)
    SHA_DEV_CTX pmk_ctx;

    // Thread ID
    const int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    //ORIGINAL
    // Is ID out of range?
    //if ( id >= max_num )
      //  return;

    // First round's hash (for PMK1) is stored in e1
    COPY_DEVCTX( prev_ctx , inbuffer[id].e1 );
    COPY_DEVCTX( pmk_ctx , prev_ctx );

    // PMK Part 1 (20 bytes): Finish the remaining 4095 rounds
    for ( i = 1 ; i <= 4095 ; i++ ) {

        sha1_process ( &inbuffer[id].ctx_ipad , &prev_ctx );
        sha1_process ( &inbuffer[id].ctx_opad , &prev_ctx );

        // Keep XORing all the rounds
        pmk_ctx.h0 ^= prev_ctx.h0;
        pmk_ctx.h1 ^= prev_ctx.h1;
        pmk_ctx.h2 ^= prev_ctx.h2;
        pmk_ctx.h3 ^= prev_ctx.h3;
        pmk_ctx.h4 ^= prev_ctx.h4;
    }

    // Store PMK Part 1 in the output buffer
    COPY_DEVCTX( outbuffer[id].pmk1 , pmk_ctx );

    // First round's hash (for PMK2) is stored in e2
    COPY_DEVCTX( prev_ctx , inbuffer[id].e2 );
    COPY_DEVCTX( pmk_ctx , prev_ctx );
    
    // PMK Part 2 (12 bytes): Finish the remaining 4095 rounds
    for ( i = 1 ; i <= 4095 ; i++ ) {
    
        sha1_process ( &inbuffer[id].ctx_ipad , &prev_ctx );
        sha1_process ( &inbuffer[id].ctx_opad , &prev_ctx );
        
        // Keep XORing all the rounds
        pmk_ctx.h0 ^= prev_ctx.h0;
        pmk_ctx.h1 ^= prev_ctx.h1;
        pmk_ctx.h2 ^= prev_ctx.h2; 
        pmk_ctx.h3 ^= prev_ctx.h3;
        pmk_ctx.h4 ^= prev_ctx.h4;
    }
    
    // Store PMK Part 2 in the output buffer
    COPY_DEVCTX( outbuffer[id].pmk2 , pmk_ctx );
}

// Function that triggers the GPU Kernel (Device Thread)
void invoke_gpu_kernel ( int blocksPerGrid , int threadsPerBlock , 
                         kernel_input_buffer *inbuffer , kernel_output_buffer *outbuffer , int max_num ) {
    crack_gpu_kernel<<<blocksPerGrid, threadsPerBlock>>>( inbuffer , outbuffer , max_num );
}
