
// Endianess, init values, speed report, finish report, spin lock on GPUs????, error checking
// Optimization - Verifying MIC some parts are not required or can be done beforehand
// TODOs
// Check speed by not reporting finish, 
// turn off cpus and see if gpu working, etc.
// Set the THREADS_PER_BLOCK and BATCH_SIZE accordingly for different GPUs
// Output total time to crack
// GPU asynch calls what to do??

#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sched.h>
#include <pthread.h>
#include <sys/time.h>

#include <openssl/hmac.h>
#include <openssl/sha.h>

// Helper functions common to CUDA SDK samples
#include "headers/sdkHelper.h"
#include "headers/shrQATest.h"
#include "headers/shrUtils.h"

#include "headers/cpu-crack.h"
#include "headers/gpu-crack.h"

#ifndef uchar
#define uchar unsigned char
#endif

#ifndef uint
#define uint unsigned int
#endif



// These are CUDA Helper functions

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
	if(hipSuccess != err)
	{
		fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
		exit(-1);        
	}
}

// This will output the proper error string when calling hipGetLastError
#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line )
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
				file, line, errorMessage, (int)err, hipGetErrorString( err ) );
		exit(-1);
	}
}

// End of CUDA Helper Functions

//GLOBALS
//Connection to MySQL DB
extern MYSQL* MySQLConnection[NUM_DB_CONNECTIONS];
//Stores the list of passwords we queried for
//char **passwordList;
extern unsigned long keys;
char* final_key; 
//verbosity flag
extern int vflag;
// Returns the number of GPUs (possibly 0)
int 
num_of_gpus() {
	int deviceCount;
	checkCudaErrors ( hipGetDeviceCount ( &deviceCount ) );
	return deviceCount;
}

// Precompute the IKeypad, OKeypad and 1st Round hashes
inline 
void 
precompute ( const char *key, const char *essid_pre , kernel_input_buffer *gpu_input ) {

	// Key length
	int keyLength = strlen ( key );
	// ESSID length
	int essidLength = strlen ( essid_pre );

	// ESSID (extra 4 bytes for the '\1' and '\2' during calculation)
	uchar essid[32 + 4];
	memset ( essid , 0 , sizeof ( essid ) );
	memcpy ( essid , essid_pre , essidLength );

	// 64 byte buffer used in SHA-1
	uchar buffer[64];

	//Unspecific Loop variables
	int i;


	// Current hash state
	SHA_CTX ctx_pad;

	// Calculate the Keypad initial hashes
	// IKeypad
	memcpy ( buffer , key , keyLength );
	memset ( buffer + keyLength , 0 , sizeof ( buffer ) - keyLength );
	for ( i = 0 ; i < sizeof ( buffer ) / sizeof ( uint32_t ) ; ++i )
		( (uint32_t*) buffer )[i] ^= 0x36363636;
	SHA1_Init ( &ctx_pad );
	SHA1_Update ( &ctx_pad , buffer , sizeof ( buffer ) );
	COPY_DEVCTX( gpu_input -> ctx_ipad , ctx_pad );

	// OKeypad
	for ( i = 0 ; i < sizeof ( buffer ) / sizeof ( uint32_t ) ; ++i )
		( (uint32_t*) buffer )[i] ^= 0x6A6A6A6A;
	SHA1_Init ( &ctx_pad );
	SHA1_Update ( &ctx_pad , buffer , sizeof ( buffer ) );
	COPY_DEVCTX( gpu_input -> ctx_opad , ctx_pad );

	// 1st Round hashes
	uchar temp[20];
	// ESSID '\1'
	essid [ essidLength + 4 - 1 ] = '\1';
	HMAC ( EVP_sha1() , key , keyLength , essid , essidLength + 4 , temp , NULL );
	GET_BE( gpu_input -> e1.h0 , temp , 0 );
	GET_BE( gpu_input -> e1.h1 , temp , 4 );
	GET_BE( gpu_input -> e1.h2 , temp , 8 );
	GET_BE( gpu_input -> e1.h3 , temp , 12 );
	GET_BE( gpu_input -> e1.h4 , temp , 16 );

	// ESSID '\2'
	essid [ essidLength + 4 - 1 ] = '\2';
	HMAC ( EVP_sha1() , key , keyLength , essid , essidLength + 4 , temp , NULL );
	GET_BE( gpu_input -> e2.h0 , temp , 0 );
	GET_BE( gpu_input -> e2.h1 , temp , 4 );
	GET_BE( gpu_input -> e2.h2 , temp , 8 );
	GET_BE( gpu_input -> e2.h3 , temp , 12 );
	GET_BE( gpu_input -> e2.h4 , temp , 16 );
}
// This is where the information from the .cap file is used
// Check if the key was found, by verifying the MIC
inline
int 
is_key_found ( const kernel_output_buffer *gpu_output , const wpa_hdsk *phdsk ) {

	// PKE and PTK used in MIC calculation
	uchar pke[100];
	uchar ptk[80];
	uchar mic[20];
	// Extract the 32 byte PMK from the gpu output buffer
	uchar pmk[32];
	PUT_BE( gpu_output -> pmk1.h0 , pmk , 0 );
	PUT_BE( gpu_output -> pmk1.h1 , pmk , 4 );
	PUT_BE( gpu_output -> pmk1.h2 , pmk , 8 );
	PUT_BE( gpu_output -> pmk1.h3 , pmk , 12 );
	PUT_BE( gpu_output -> pmk1.h4 , pmk , 16 );
	PUT_BE( gpu_output -> pmk2.h0 , pmk , 20 );
	PUT_BE( gpu_output -> pmk2.h1 , pmk , 24 );
	PUT_BE( gpu_output -> pmk2.h2 , pmk , 28 );

	// Loop Variable
	int i = 0;

	// Construct the key expansion buffer
	memcpy ( pke , "Pairwise key expansion" , 23 );
	// Add the MACs
	if ( memcmp ( phdsk -> smac , phdsk -> amac , 6 ) < 0 ) {
		memcpy ( pke + 23 , phdsk -> smac , 6 );
		memcpy ( pke + 29 , phdsk -> amac , 6 );
	}
	else {
		memcpy ( pke + 23 , phdsk -> amac , 6 );
		memcpy ( pke + 29 , phdsk -> smac , 6 );
	}
	// Add the Nonces
	if ( memcmp ( phdsk -> snonce , phdsk -> anonce , 32 ) < 0 ) {
		memcpy ( pke + 35 , phdsk -> snonce , 32 );
		memcpy ( pke + 67 , phdsk -> anonce , 32 );
	}
	else {
		memcpy ( pke + 35 , phdsk -> anonce , 32 );
		memcpy ( pke + 67 , phdsk -> snonce , 32 );
	}

	// Calculate the PTK
	for ( i = 0 ; i < 4 ; i++ ) {
		pke[99] = i;
		HMAC ( EVP_sha1() , pmk , 32 , pke , 100 , ptk + i * 20 , NULL );
	}

	// Calculate the MIC
	if ( phdsk -> keyver == 1 )
		HMAC ( EVP_md5() , ptk , 16 , phdsk -> eapol , phdsk -> eapol_size , mic , NULL );
	else
		HMAC ( EVP_sha1() , ptk , 16 , phdsk -> eapol , phdsk -> eapol_size , mic , NULL );

	// Check if MIC agrees
	if ( memcmp ( mic , phdsk -> keymic , 16 ) == 0 )
	{
		printf("FOUND!\n");
		return 1;
	}
	// Return 0 if not found, 1 if found
	return 0;
}
/**
 * Used to get the password given a particular offset
 * Used to get the correct password immediately before exiting
 */
int get_password(int db_connector_index, int gpu_num,unsigned long offset)
{

	//connection to DB
	MYSQL_RES      *mysqlResult = NULL;
	MYSQL_ROW       mysqlRow;
	//unsigned int numRows;
	//unsigned int numFields;
	int mysqlStatus = 0;
	//for the query
	char query[QUERY_BUFFER];

	// --------------------------------------------------------------------
	// Perform a SQL SELECT and retrieve data
	// There should not be a terminating ';'
	sprintf(query, "SELECT %s FROM %s LIMIT 1 OFFSET %lu",COLUMN_NAME,TABLE_NAME,offset);
	// printf("Query is: %s\n",query);
	//printf("Range start is : %lu\n",range.start);

	mysqlStatus = mysql_query(MySQLConnection[db_connector_index],query);
	if (mysqlStatus)
	{
		fprintf(stderr,"Unable to connect to get the password at %lu, quitting\n",offset);
		return 0; 
	}
	else
	{
		mysqlResult = mysql_store_result(MySQLConnection[db_connector_index]); // Get the Result Set
	}
	if(!mysqlResult)
	{
		fprintf(stderr,"Result set is empty");
		mysql_close(MySQLConnection[db_connector_index]);
		return 0;
	}
	if(mysqlRow = mysql_fetch_row(mysqlResult)) // row pointer in the result set
	{
		printf("%s\n",mysqlRow[0]);
		memcpy ( final_key , mysqlRow[0] , strlen ( mysqlRow[0] ) ); 
		return 1;
	}
	return 0;
}
//make query and place potential passwords into the already allocated space
unsigned int query_and_fill(int db_connector_index, int gpu_id,char* essid,unsigned long start_index,kernel_input_buffer *gpu_input)
{

	//connection to DB
	MYSQL_RES      *mysqlResult = NULL;
	MYSQL_ROW       mysqlRow;
	//unsigned int numRows;
	//unsigned int numFields;
	int mysqlStatus = 0;
	//for the query
	char query[QUERY_BUFFER];


	unsigned int password_index;
	//how many passwords we saved
	//set back to 0	
	password_index = 0;
	// --------------------------------------------------------------------
	// Perform a SQL SELECT and retrieve data
	// There should not be a terminating ';'
	sprintf(query, "SELECT %s FROM %s LIMIT %d OFFSET %lu",COLUMN_NAME,TABLE_NAME,PWD_BATCH_SIZE_GPU,start_index);
	if(vflag)
	{
		printf("GPU Query is %s\n",query);
	}
	//printf("Range start is : %lu\n",range.start);

	mysqlStatus = mysql_query(MySQLConnection[db_connector_index],query);
	if (mysqlStatus)
	{
		fprintf(stderr,"GPU Thread: MySQL Error:\nQuitting");
		return 0;
	}
	else
	{
		mysqlResult = mysql_store_result(MySQLConnection[db_connector_index]); // Get the Result Set
	}
	/* if (mysqlResult)  // there are rows
	   {
	// # of rows in the result set
	numRows = mysql_num_rows(mysqlResult);

	// Returns the number of columns in a result set specified
	numFields = mysql_num_fields(mysqlResult);

	//printf("Number of rows=%u  Number of fields=%u \n",numRows,numFields);
	}*/
	if(!mysqlResult)
	{
		fprintf(stderr,"Result set is empty");
		return 0;
	}
	while(mysqlRow = mysql_fetch_row(mysqlResult)) // row pointer in the result set
	{
		//IF we just kept track of the index of where the password was found,
		//we THINK we could replace the two lines below with the following:
		//no longer necessary to keep track of password, we just track index
		// Precompute the iKeypads, oKeypads and 1st Round Hashes
		//precompute(mysqlRow[0], essid, & gpu_input[password_index]);
		precompute(mysqlRow[0], essid, & gpu_input[ (gpu_id*PWD_BATCH_SIZE_GPU)+ password_index]);


		// Count the total number of keys
		password_index++;
	}
	if(vflag)
	{
		printf("GPU got %u number of rows\n",password_index);
	}
	return password_index;
}
void cleanUp(int cpu_num,int gpu_num)
{

	printf("GPU thread: closing DB connection\n");
	mysql_close(MySQLConnection[cpu_num]);
	printf("Tested ");
	printf("%lu",keys);
	printf(" keys total.\n");
	printf("GPU thread exiting\n");

}
// GPU Crack Host thread
void* 
crack_gpu_thread ( void *arg ) {

	// ESSID
	char essid[32];

	//ORIGINAL
	// Password (key) in string format
	//char key[128];
	//memset ( key , 0 , sizeof ( key ) );

	// Params passed in arguments
	ck_td_struct* ck_td_arg = (ck_td_struct*)arg;
	wpa_hdsk* phdsk = ck_td_arg->phdsk;
	int cpu_num = ck_td_arg->cpu_core_id;
	int gpu_num = ck_td_arg->gpu_core_id;
	if ( gpu_num == 0 ) // GPU not used
	{
		printf("No GPUs found\n");
		return NULL;
	}   
	float* calc_speed = ck_td_arg->calc_speed;
	final_key = ck_td_arg->final_key;
	char* final_key_flag = ck_td_arg->final_key_flag;
	memset(essid, 0, sizeof(essid));
	memcpy(essid, ck_td_arg->essid, 32);

	// Normal (unspecific) Loop Variable
	int i = 0;

	//iterates through the gpu_num in loops
	int gpu_iter;

	//index variable for the password range
	unsigned int password_total;

	// For calculating the PMK/sec speed
	struct timeval tprev;
	struct timeval tnow;

	gettimeofday (&tprev, NULL );
	// Password range for each GPU
	pwd_range *range;
	range = (pwd_range*) malloc ( sizeof ( pwd_range )*gpu_num);

	//time that has passed
	float total_time =0.00;
	/*
	//allocate password space

	printf("Allocating space for %d passwords of length %d\n",(gpu_num * PWD_BATCH_SIZE_GPU),LONGEST_PASSWORD);
	passwordList = (char**) calloc( (gpu_num * PWD_BATCH_SIZE_GPU),  sizeof(char*));

	for(i = 0; i <gpu_num*PWD_BATCH_SIZE_GPU ; i++) {
	passwordList[i] = (char*) calloc(LONGEST_PASSWORD, sizeof(char));

	}

	 */
	// Number of working GPUs
	int gpu_working = 0;

	printf("%d threads per block\n",THREADS_PER_BLOCK);
	printf("%d blocks per grid\n",( PWD_BATCH_SIZE_GPU + THREADS_PER_BLOCK - 1 ) / THREADS_PER_BLOCK);
	// Input Buffer of the GPUs
	kernel_input_buffer *gpu_input = (kernel_input_buffer*) malloc ( sizeof ( kernel_input_buffer ) * PWD_BATCH_SIZE_GPU * gpu_num );
	// Output Buffer of the GPUs
	kernel_output_buffer *gpu_output = (kernel_output_buffer*) malloc ( sizeof ( kernel_output_buffer ) * PWD_BATCH_SIZE_GPU * gpu_num );

	// Buffers in the GPU Memory space
	kernel_input_buffer **device_input;
	kernel_output_buffer **device_output;

	if(vflag==1)
	{
		printf("malloc called for input and output buffers\n");
	}
	device_input = (kernel_input_buffer**) malloc ( sizeof ( kernel_input_buffer* ) * gpu_num );
	device_output = (kernel_output_buffer**) malloc ( sizeof ( kernel_output_buffer* ) * gpu_num );

	// Allocate device memory beforehand itself, we can reuse it again and again
	int devMemSize;
	for ( gpu_iter = 0 ; gpu_iter < gpu_num ; ++gpu_iter ) {
		if(vflag==1)
		{
			printf("cuda malloc called for gpu %d of %d\n",gpu_iter,gpu_num);
		}
		checkCudaErrors ( hipSetDevice(gpu_iter) );
		devMemSize = sizeof ( kernel_input_buffer ) * PWD_BATCH_SIZE_GPU;
		checkCudaErrors ( hipMalloc ( (void**) &device_input[gpu_iter] , devMemSize ) );
		devMemSize = sizeof ( kernel_output_buffer ) * PWD_BATCH_SIZE_GPU;
		checkCudaErrors ( hipMalloc ( (void**) &device_output[gpu_iter] , devMemSize ) );
	}

	// Repeatedly get password ranges to dispatch to the GPU(s)
	while ( 1 ) {

		// Get the password range for each gpu
		gpu_working = 0;
		//original
		for ( gpu_iter = 0 ; gpu_iter < gpu_num ; ++gpu_iter ) {

			range[gpu_iter] = fetch_pwd ( 'g' , NULL,NULL);
			++gpu_working;
			if ( range[gpu_iter].start == 0.5 )
			{
				fprintf(stderr,"starting range is invalid for GPU, quitting\n");
				// Tell main thread we are terminating
				gpu_working=0;
				goto stop;
			}

			//for the number of GPUs working
			password_total=query_and_fill(cpu_num,gpu_iter,essid,range[gpu_iter].start,gpu_input);
			//this will happen when query does not return anymore rows
			if(password_total == 0)
			{
				printf("read 0 passwords from %s in %s\nQuitting!\n",TABLE_NAME,DB_NAME);
				gpu_working=0;
				goto stop;
			}
			//password_total should be the number of passwords in the range, meaning we looped through all of them and precomputed them
			keys+=password_total;
		}	
		/*
		   if(vflag==1)
		   {
		   printf("GPU id %d offset is: %lu\n",gpu_iter,range[0].start);
		   }*/

		//	}
		//printf("%d\n",gpu_working);
		// Check if password range is over
		if ( gpu_working <= 0 ) {

			// Tell main thread we are terminating
			calc_speed[ cpu_num ] = -1;

			// Free resources
			free ( range );
			free ( gpu_input );
			free ( gpu_output );
			for ( gpu_iter = 0 ; gpu_iter < gpu_num ; ++gpu_iter ) {
				checkCudaErrors ( hipSetDevice (gpu_iter) );
				checkCudaErrors ( hipFree ( (void*) device_input[gpu_iter] ) );
				checkCudaErrors ( hipFree ( (void*) device_output[gpu_iter] ) );
			}
			free ( device_input );
			free ( device_output );
			printf("\nNo GPUs working, quitting\n");
			goto stop;
		}



		// Start time of the computation (including memory transfers Host mem <==> Device mem)
		//	gettimeofday ( &tprev , NULL );

		//printf("Total number of passwords checked: %lu in \n",keys);
		//for each unique password in our range
		//  for (i = range[gpu_iter].start; i < range[gpu_iter].end ; i++ ) 
		//this seems right,
		//FIX THE REST...later

		/*
		   THIS IS NOW BEING DONE IN QUERY_AND_FILL

		   for(i=0;i<total_num_passwords;i++)
		   {
		//ORIGINAL
		// Convert the key from digit to string
		//             sprintf ( key , "%08lu" , cur_key_digit );

		//strcpy(key,passwordList[i]);
		//SUNJAY
		//printf("%d) GPU Password: %s\n",i,key);	
		// Calculate the Kernel input buffer values for this key
		//precompute (passwordList[i], essid , & gpu_input[ ( gpu_iter * PWD_BATCH_SIZE_GPU ) + (i - range[gpu_iter].start ) ] );
		 ******
		//ORIGINAL 
		// Calculate the Kernel input buffer values for this key
		precompute ( key , essid , & gpu_input[ ( i * PWD_BATCH_SIZE_GPU ) + ( cur_key_digit - range[i].start ) ] );
		 *****
		//precompute (passwordList[i], essid , & gpu_input[i]);


		// Count the total number of keys
		++num_keys;
		// if(num_keys%100==0)
		//  printf("Total number of passwords we precomputed: %d\n",num_keys);
		}*/
		// }
		//printf("Total number of passwords read is %d\n",total_num_passwords);

		// if(num_keys%PWD_BATCH_SIZE_GPU==0)
		// {
		//  printf("Total number of passwords we precomputed: %d\n",num_keys);
		// }
/*if(num_keys!=total_num_passwords)
  {
  printf("We did NOT precompute all the passwords\n");
  printf("Total number of passwords we got from the DB: %d\n",total_num_passwords);
  printf("Total number of passwords we precomputed: %d\n",num_keys);
  }*/
// Now let the GPUs do the work
/*

 * We need to be careful here. The calls to CUDA runtime API are asynchronous.
 * The CUDA manual doesn't really explain it properly :P
 * Here's the deal: (http://forums.nvidia.com/index.php?showtopic=175073)
 * 1. Copies from Host to Device are asynchchronous if data size <= 64 kb, otherwise they are synchronous
 * 2. Kernel calls are ALWAYS asynchronous
 * 3. Copies from Device to Host are ALWAYS synchronous (otherwise we cannot use the output buffer after the call!!)
 * 
 * Here the data size is 10000 * sizeof ( input buffer ) == 10000 * 80 = 800000 bytes = approx 800 kb
 * So, the call to copy from HtoD is Synch. Can't do anything about that :P
 * 
 * But, the copying of data from Device to Host (Synch) should be in a SEPARATE FOR LOOP. Otherwise, we cannot
 * dispatch work to the second GPU until the first GPU finishes computation :P
 */

if(vflag)
{
	printf("There are %d GPUs working\n",gpu_working);
}
for ( gpu_iter = 0 ; gpu_iter < gpu_working ; ++gpu_iter ) 
{
	// Set the GPU Device we are currently dispatching work to crack
	checkCudaErrors ( hipSetDevice ( gpu_iter ) );

	// Copy the Input buffers from the Host to Device (GPU) Memory
	devMemSize = sizeof ( kernel_input_buffer ) * PWD_BATCH_SIZE_GPU;
	checkCudaErrors ( hipMemcpy ( device_input[gpu_iter] , gpu_input + (gpu_iter * PWD_BATCH_SIZE_GPU) , devMemSize , hipMemcpyHostToDevice ) );

	// Calculate the PMKs using GPU
	//int max_num = range[gpu_iter].end - range[gpu_iter].start + 1;
	//GPU ALWAYS HAS MAX NUMBER OF THREADS, regardless of number of passwords
	int blocksPerGrid = (PWD_BATCH_SIZE_GPU + THREADS_PER_BLOCK - 1 ) / THREADS_PER_BLOCK;

	invoke_gpu_kernel ( blocksPerGrid , THREADS_PER_BLOCK , device_input[gpu_iter] , device_output[gpu_iter] , PWD_BATCH_SIZE_GPU);
	getLastCudaError ( "Kernel launch failure!!" );
}

// Copy the Output buffers to the Host from Device (GPU) Memory in SEPARATE FOR LOOP
for ( gpu_iter = 0 ; gpu_iter < gpu_working ; ++gpu_iter ) {

	// Set the GPU Device we are currently dispatching work to crack
	checkCudaErrors ( hipSetDevice ( gpu_iter ) );

	// Copy the Output buffers to the Host from Device (GPU) Memory
	devMemSize = sizeof ( kernel_output_buffer ) * PWD_BATCH_SIZE_GPU;
	checkCudaErrors ( hipMemcpy ( gpu_output + (gpu_iter * PWD_BATCH_SIZE_GPU) , device_output[gpu_iter] , devMemSize , hipMemcpyDeviceToHost ) );
}

// Check if the key (password) was found
for ( gpu_iter = 0 ; gpu_iter < gpu_working ; ++gpu_iter ) {
	//ORIGINAL
	//loop through all passwords for a particular gpu
	//for (i = range[gpu_iter].start; i <= range[gpu_iter].end; ++i ) {
	//printf("Checking if passwords from %d to %d are the password\n",(gpu_iter*PWD_BATCH_SIZE_GPU),(PWD_BATCH_SIZE_GPU*(gpu_iter+1)-1));
	for(i=gpu_iter*PWD_BATCH_SIZE_GPU;i<PWD_BATCH_SIZE_GPU*(gpu_iter+1);i++){	
		//for(i=gpu_iter*PWD_BATCH_SIZE_GPU;i<PWD_BATCH_SIZE_GPU+(gpu_iter*PWD_BATCH_SIZE_GPU);i++){
		//printf("checking %d for key\n",i);
		// Verify the MIC
		//ORIGINAL
		//if ( is_key_found ( & gpu_output[ ( gpu_iter * PWD_BATCH_SIZE_GPU ) + ( i - range[gpu_iter].start ) ] , phdsk ) ) {
		if ( is_key_found (&gpu_output[i] , phdsk) ) {
			*final_key_flag = 1;

			//printf("GPU found the key\nPassword: %s\n",passwordList[i]);
			unsigned int password_found_index=range[gpu_iter].start+(i-(gpu_iter*PWD_BATCH_SIZE_GPU));
			printf("GPU found the key\nPassword is at index %u\n",password_found_index);
			if(get_password(cpu_num,gpu_num,password_found_index))
			{
				printf("Successfully looked up password.\n");
			}
			else
			{
				printf("Error in get_password() function\n");
			}

			// !!!!! We found the key !!!!!

			// End time of computation (including memory transfers Host mem <==> Device mem)

			// Report speed to main thread
			//calc_speed[ cpu_num ] = (float) num_keys / ( tnow.tv_sec - tprev.tv_sec + ( tnow.tv_usec - tprev.tv_usec ) * 0.000001F );

			// Sleep a little so that the main thread will read the speed
			// sleep ( 1 );

			// Convert the key from digit to string
			//ORIGINAL
			//sprintf ( key , "%08lu" , cur_key_digit );
			// Report the key to the main thread
			//ORIGINALLY WAS NOT COMMENTED OUT

			//memcpy ( final_key , passwordList[i] , strlen (passwordList[i] ) );

			// Tell main thread we are terminating
			// calc_speed[ cpu_num ] = -1;

			// Free resources
			free ( range );
			free ( gpu_input );
			free ( gpu_output );

			int ii;
			for ( ii = 0 ; ii < gpu_num ; ++ii ) {
				checkCudaErrors ( hipSetDevice ( ii ) );
				checkCudaErrors ( hipFree ( (void*) device_input[ii] ) );
				checkCudaErrors ( hipFree ( (void*) device_output[ii] ) );
			}
			free ( device_input );
			free ( device_output );
			goto stop;
			//closes if key is found
		}
		//closes for loop for iterating through generated pmks
	}
	//closes for loop for passwords generated by a single gpu, incrementing by GPU_PWD_BATCH_SIZE
	}

	gettimeofday ( &tnow , NULL );
	total_time = tnow.tv_sec - tprev.tv_sec + ( tnow.tv_usec - tprev.tv_usec ) * 0.000001F;
	printf ( "Tested %lu passwords in %.2f seconds.\n" ,keys,total_time);
	// End time of computation (including memory transfers Host mem <==> Device mem)
	//gettimeofday ( &tnow , NULL );

	// Report speed to main thread
	// calc_speed[ cpu_num ] = (float) num_keys / ( tnow.tv_sec - tprev.tv_sec + ( tnow.tv_usec - tprev.tv_usec ) * 0.000001F );
	}
stop: 
	// Tell main thread we are terminating
	calc_speed[ cpu_num ] = -1;
	gettimeofday ( &tnow , NULL );
	cleanUp(cpu_num,gpu_num);
	//hipDeviceReset();
	return NULL;
}
