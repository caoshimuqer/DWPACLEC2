
// Endianess, init values, speed report, finish report, spin lock on GPUs????, error checking
// Optimization - Verifying MIC some parts are not required or can be done beforehand
// TODOs
// Check speed by not reporting finish, turn off cpus and see if gpu working, etc.
// Set the THREADS_PER_BLOCK and BATCH_SIZE accordingly for different GPUs
// Output total time to crack
// GPU asynch calls what to do??

#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sched.h>
#include <pthread.h>
#include <sys/time.h>

#include <openssl/hmac.h>
#include <openssl/sha.h>

// Helper functions common to CUDA SDK samples
#include "sdkHelper.h"
#include "shrQATest.h"
#include "shrUtils.h"

#include "cpu-crack.h"
#include "gpu-crack.h"

#ifndef uchar
#define uchar unsigned char
#endif

#ifndef uint
#define uint unsigned int
#endif

// These are CUDA Helper functions

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
    if(hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
        exit(-1);        
    }
}

// This will output the proper error string when calling hipGetLastError
#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line )
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
        file, line, errorMessage, (int)err, hipGetErrorString( err ) );
        exit(-1);
    }
}

// End of CUDA Helper Functions

// Returns the number of GPUs (possibly 0)
int 
num_of_gpus() {
    int deviceCount;
    checkCudaErrors ( hipGetDeviceCount ( &deviceCount ) );
    return deviceCount;
}

// Precompute the IKeypad, OKeypad and 1st Round hashes
inline 
void 
precompute ( const char *key , const char *essid_pre , kernel_input_buffer *gpu_input ) {

    // Key length
    int keyLength = strlen ( key );
    // ESSID length
    int essidLength = strlen ( essid_pre );
    
    // ESSID (extra 4 bytes for the '\1' and '\2' during calculation)
    uchar essid[32 + 4];
    memset ( essid , 0 , sizeof ( essid ) );
    memcpy ( essid , essid_pre , essidLength );
    
    // 64 byte buffer used in SHA-1
    uchar buffer[64];
    
    // Loop variables
    int i;
    
    // Current hash state
    SHA_CTX ctx_pad;
    
    // Calculate the Keypad initial hashes
    // IKeypad
    memcpy ( buffer , key , keyLength );
    memset ( buffer + keyLength , 0 , sizeof ( buffer ) - keyLength );
    for ( i = 0 ; i < sizeof ( buffer ) / sizeof ( uint32_t ) ; ++i )
        ( (uint32_t*) buffer )[i] ^= 0x36363636;
    SHA1_Init ( &ctx_pad );
    SHA1_Update ( &ctx_pad , buffer , sizeof ( buffer ) );
    COPY_DEVCTX( gpu_input -> ctx_ipad , ctx_pad );
    
    // OKeypad
    for ( i = 0 ; i < sizeof ( buffer ) / sizeof ( uint32_t ) ; ++i )
        ( (uint32_t*) buffer )[i] ^= 0x6A6A6A6A;
    SHA1_Init ( &ctx_pad );
    SHA1_Update ( &ctx_pad , buffer , sizeof ( buffer ) );
    COPY_DEVCTX( gpu_input -> ctx_opad , ctx_pad );
    
    // 1st Round hashes
    uchar temp[20];
    // ESSID '\1'
    essid [ essidLength + 4 - 1 ] = '\1';
    HMAC ( EVP_sha1() , key , keyLength , essid , essidLength + 4 , temp , NULL );
    GET_BE( gpu_input -> e1.h0 , temp , 0 );
    GET_BE( gpu_input -> e1.h1 , temp , 4 );
    GET_BE( gpu_input -> e1.h2 , temp , 8 );
    GET_BE( gpu_input -> e1.h3 , temp , 12 );
    GET_BE( gpu_input -> e1.h4 , temp , 16 );
    
    // ESSID '\2'
    essid [ essidLength + 4 - 1 ] = '\2';
    HMAC ( EVP_sha1() , key , keyLength , essid , essidLength + 4 , temp , NULL );
    GET_BE( gpu_input -> e2.h0 , temp , 0 );
    GET_BE( gpu_input -> e2.h1 , temp , 4 );
    GET_BE( gpu_input -> e2.h2 , temp , 8 );
    GET_BE( gpu_input -> e2.h3 , temp , 12 );
    GET_BE( gpu_input -> e2.h4 , temp , 16 );
}

int areWeDone()
{

}
// This is where the information from the .cap file is used
// Check if the key was found, by verifying the MIC
inline
int 
is_key_found ( const kernel_output_buffer *gpu_output , const wpa_hdsk *phdsk ) {
    
    // PKE and PTK used in MIC calculation
    uchar pke[100];
    uchar ptk[80];
    uchar mic[20];
    
    // Extract the 32 byte PMK from the gpu output buffer
    uchar pmk[32];
    PUT_BE( gpu_output -> pmk1.h0 , pmk , 0 );
    PUT_BE( gpu_output -> pmk1.h1 , pmk , 4 );
    PUT_BE( gpu_output -> pmk1.h2 , pmk , 8 );
    PUT_BE( gpu_output -> pmk1.h3 , pmk , 12 );
    PUT_BE( gpu_output -> pmk1.h4 , pmk , 16 );
    PUT_BE( gpu_output -> pmk2.h0 , pmk , 20 );
    PUT_BE( gpu_output -> pmk2.h1 , pmk , 24 );
    PUT_BE( gpu_output -> pmk2.h2 , pmk , 28 );

    // Loop Variable
    int i = 0;

    // Construct the key expansion buffer
    memcpy ( pke , "Pairwise key expansion" , 23 );
    // Add the MACs
    if ( memcmp ( phdsk -> smac , phdsk -> amac , 6 ) < 0 ) {
        memcpy ( pke + 23 , phdsk -> smac , 6 );
        memcpy ( pke + 29 , phdsk -> amac , 6 );
    }
    else {
        memcpy ( pke + 23 , phdsk -> amac , 6 );
        memcpy ( pke + 29 , phdsk -> smac , 6 );
    }
    // Add the Nonces
    if ( memcmp ( phdsk -> snonce , phdsk -> anonce , 32 ) < 0 ) {
        memcpy ( pke + 35 , phdsk -> snonce , 32 );
        memcpy ( pke + 67 , phdsk -> anonce , 32 );
    }
    else {
        memcpy ( pke + 35 , phdsk -> anonce , 32 );
        memcpy ( pke + 67 , phdsk -> snonce , 32 );
    }

    // Calculate the PTK
    for ( i = 0 ; i < 4 ; i++ ) {
        pke[99] = i;
        HMAC ( EVP_sha1() , pmk , 32 , pke , 100 , ptk + i * 20 , NULL );
    }

    // Calculate the MIC
    if ( phdsk -> keyver == 1 )
        HMAC ( EVP_md5() , ptk , 16 , phdsk -> eapol , phdsk -> eapol_size , mic , NULL );
    else
        HMAC ( EVP_sha1() , ptk , 16 , phdsk -> eapol , phdsk -> eapol_size , mic , NULL );

    // Check if MIC agrees
    if ( memcmp ( mic , phdsk -> keymic , 16 ) == 0 )
        return 1;

    // Return 0 if not found, 1 if found
    return 0;
}

// GPU Crack Host thread
void* 
crack_gpu_thread ( void *arg ) {

    // ESSID
    char essid[32];
    
    // Password (key) in string format
    char key[128];
    memset ( key , 0 , sizeof ( key ) );

    // Params passed in arguments
    ck_td_struct* ck_td_arg = (ck_td_struct*)arg;
    wpa_hdsk* phdsk = ck_td_arg->phdsk;
    int cpu_num = ck_td_arg->cpu_core_id;
    int gpu_num = ck_td_arg->gpu_core_id;
    if ( gpu_num == 0 ) // GPU not used
        return NULL;
    float* calc_speed = ck_td_arg->calc_speed;
    char* final_key = ck_td_arg->final_key;
    char* final_key_flag = ck_td_arg->final_key_flag;
    memset(essid, 0, sizeof(essid));
    memcpy(essid, ck_td_arg->essid, 32);
  
    // Normal Loop Variable
    int i = 0;
    
    // Loop variable for the password range
    unsigned long cur_key_digit = 0;

    // For calculating the PMK/sec speed
    struct timeval tprev;
    struct timeval tnow;

    // Password range for each GPU
    pwd_range *range;
    range = (pwd_range*) malloc ( sizeof ( pwd_range ) * gpu_num );
    
    // Number of working GPUs
    int gpu_working = 0;
    // Number of PMKs being computed in one dispatch
    int num_keys = 0;
    
    // Input Buffer of the GPUs
    kernel_input_buffer *gpu_input = (kernel_input_buffer*) malloc ( sizeof ( kernel_input_buffer ) * PWD_BATCH_SIZE_GPU * gpu_num );
    // Output Buffer of the GPUs
    kernel_output_buffer *gpu_output = (kernel_output_buffer*) malloc ( sizeof ( kernel_output_buffer ) * PWD_BATCH_SIZE_GPU * gpu_num );
    
    // Buffers in the GPU Memory space
    kernel_input_buffer **device_input;
    kernel_output_buffer **device_output;
    device_input = (kernel_input_buffer**) malloc ( sizeof ( kernel_input_buffer* ) * gpu_num );
    device_output = (kernel_output_buffer**) malloc ( sizeof ( kernel_output_buffer* ) * gpu_num );
    
    // Allocate device memory beforehand itself, we can reuse it again and again
    int devMemSize;
    for ( i = 0 ; i < gpu_num ; ++i ) {
        checkCudaErrors ( hipSetDevice ( i ) );
        devMemSize = sizeof ( kernel_input_buffer ) * PWD_BATCH_SIZE_GPU;
        checkCudaErrors ( hipMalloc ( (void**) &device_input[i] , devMemSize ) );
        devMemSize = sizeof ( kernel_output_buffer ) * PWD_BATCH_SIZE_GPU;
        checkCudaErrors ( hipMalloc ( (void**) &device_output[i] , devMemSize ) );
    }
    
    // Repeatedly get password ranges to dispatch to the GPUs
    while ( 1 ) {

        // Get the password range for each gpu
        gpu_working = 0;
        for ( i = 0 ; i < gpu_num ; ++i ) {
            range[i] = fetch_pwd ( 'g' , NULL , NULL );
            if ( range[i].start == -1 )
                break;
            ++gpu_working;
        }
        
        // Check if password range is over
        if ( gpu_working <= 0 ) {

            // Tell main thread we are terminating
            calc_speed[ cpu_num ] = -1;
            
            // Free resources
            free ( range );
            free ( gpu_input );
            free ( gpu_output );
            for ( i = 0 ; i < gpu_num ; ++i ) {
                checkCudaErrors ( hipSetDevice ( i ) );
                checkCudaErrors ( hipFree ( (void*) device_input[i] ) );
                checkCudaErrors ( hipFree ( (void*) device_output[i] ) );
            }
            free ( device_input );
            free ( device_output );
            
            return NULL;
        }
        
        // Start time of the computation (including memory transfers Host mem <==> Device mem)
        gettimeofday ( &tprev , NULL );
        
        // Precompute the iKeypads, oKeypads and 1st Round Hashes
        num_keys = 0;
        for ( i = 0 ; i < gpu_working ; ++i ) {
            for ( cur_key_digit = range[i].start ; cur_key_digit <= range[i].end ; ++cur_key_digit ) {
                
                // Convert the key from digit to string
                sprintf ( key , "%08lu" , cur_key_digit );
               
	       //printf("GPU Password: %s\n",key);	
                // Calculate the Kernel input buffer values for this key
                precompute ( key , essid , & gpu_input[ ( i * PWD_BATCH_SIZE_GPU ) + ( cur_key_digit - range[i].start ) ] );
                
                // Count the total number of keys
                ++num_keys;
            }
        }
        
        // Now let the GPUs do the work
        /*
         * We need to be careful here. The calls to CUDA runtime API are asynchronous.
         * The CUDA manual doesn't really explain it properly :P
         * Here's the deal: (http://forums.nvidia.com/index.php?showtopic=175073)
         * 1. Copies from Host to Device are asynchchronous if data size <= 64 kb, otherwise they are synchronous
         * 2. Kernel calls are ALWAYS asynchronous
         * 3. Copies from Device to Host are ALWAYS synchronous (otherwise we cannot use the output buffer after the call!!)
         * 
         * Here the data size is 10000 * sizeof ( input buffer ) == 10000 * 80 = 800000 bytes = approx 800 kb
         * So, the call to copy from HtoD is Synch. Can't do anything about that :P
         * 
         * But, the copying of data from Device to Host (Synch) should be in a SEPARATE FOR LOOP. Otherwise, we cannot
         * dispatch work to the second GPU until the first GPU finishes computation :P
         */
        for ( i = 0 ; i < gpu_working ; ++i ) 
	{    
            // Set the GPU Device we are currently dispatching work to crack
            checkCudaErrors ( hipSetDevice ( i ) );

            // Copy the Input buffers from the Host to Device (GPU) Memory
            devMemSize = sizeof ( kernel_input_buffer ) * PWD_BATCH_SIZE_GPU;
            checkCudaErrors ( hipMemcpy ( device_input[i] , gpu_input + (i * PWD_BATCH_SIZE_GPU) , devMemSize , hipMemcpyHostToDevice ) );
            
            // Calculate the PMKs using GPU
            int max_num = range[i].end - range[i].start + 1;
            int blocksPerGrid = ( max_num + THREADS_PER_BLOCK - 1 ) / THREADS_PER_BLOCK;
            invoke_gpu_kernel ( blocksPerGrid , THREADS_PER_BLOCK , device_input[i] , device_output[i] , max_num );
            getLastCudaError ( "Kernel launch failure!!" );
        }

        // Copy the Output buffers to the Host from Device (GPU) Memory in SEPARATE FOR LOOP
        for ( i = 0 ; i < gpu_working ; ++i ) {
            
            // Set the GPU Device we are currently dispatching work to crack
            checkCudaErrors ( hipSetDevice ( i ) );
            
            // Copy the Output buffers to the Host from Device (GPU) Memory
            devMemSize = sizeof ( kernel_output_buffer ) * PWD_BATCH_SIZE_GPU;
            checkCudaErrors ( hipMemcpy ( gpu_output + (i * PWD_BATCH_SIZE_GPU) , device_output[i] , devMemSize , hipMemcpyDeviceToHost ) );
        }
        
        // Check if the key (password) was found
        for ( i = 0 ; i < gpu_working ; ++i ) {
            for ( cur_key_digit = range[i].start ; cur_key_digit <= range[i].end ; ++cur_key_digit ) {
                
                // Verify the MIC
                if ( is_key_found ( & gpu_output[ ( i * PWD_BATCH_SIZE_GPU ) + ( cur_key_digit - range[i].start ) ] , phdsk ) ) {

                    // !!!!! We found the key !!!!!
                    
                    // End time of computation (including memory transfers Host mem <==> Device mem)
                    gettimeofday ( &tnow , NULL );
        
                    // Report speed to main thread
                    calc_speed[ cpu_num ] = (float) num_keys / ( tnow.tv_sec - tprev.tv_sec + ( tnow.tv_usec - tprev.tv_usec ) * 0.000001F );
                    
                    // Sleep a little so that the main thread will read the speed
                    sleep ( 1 );
                    
                    // Convert the key from digit to string
                    sprintf ( key , "%08lu" , cur_key_digit );
                    
                    // Report the key to the main thread
                    memcpy ( final_key , key , strlen ( key ) );
                    *final_key_flag = 1;

                    // Tell main thread we are terminating
                    calc_speed[ cpu_num ] = -1;
                    
                    // Free resources
                    free ( range );
                    free ( gpu_input );
                    free ( gpu_output );
                    for ( i = 0 ; i < gpu_num ; ++i ) {
                        checkCudaErrors ( hipSetDevice ( i ) );
                        checkCudaErrors ( hipFree ( (void*) device_input[i] ) );
                        checkCudaErrors ( hipFree ( (void*) device_output[i] ) );
                    }
                    free ( device_input );
                    free ( device_output );
                    
                    return NULL;
                }
            }
        }
        
        // End time of computation (including memory transfers Host mem <==> Device mem)
        gettimeofday ( &tnow , NULL );
        
        // Report speed to main thread
        calc_speed[ cpu_num ] = (float) num_keys / ( tnow.tv_sec - tprev.tv_sec + ( tnow.tv_usec - tprev.tv_usec ) * 0.000001F );
    }

    return NULL;
}
