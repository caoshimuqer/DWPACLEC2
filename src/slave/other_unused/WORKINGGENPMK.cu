/*
 * genpmk - Generate a file with precomputed PMK's and words
 *
 * Copyright (c) 2005, Joshua Wright <jwright@hasborg.com>
 *
 * $Id: genpmk.c,v 4.1 2008/03/20 16:49:38 jwright Exp $
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License version 2 as
 * published by the Free Software Foundation. See COPYING for more
 * details.
 *
 * coWPAtty is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 */

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <sys/stat.h>
#include <unistd.h>
#include <pcap.h>
#include <signal.h>
#include <sys/types.h>
#include <fcntl.h>
#include <errno.h>

#include "common.h"
#include "genpmk.h"
#include "sha1.cu"

#define MAGIC_NUMBER 1800

#define PROGNAME "genpmk"
#define VER "1.1"

/* Globals */
int sig = 0;			/* Used for handling signals */
char *words;

/* Prototypes */
void usage(char *message);
int nextword(char *word, FILE * fp);

void usage(char *message)
{

	if (strlen(message) > 0) {
		printf("%s: %s\n", PROGNAME, message);
	}

	printf("Usage: %s [options]\n", PROGNAME);
	printf("\n"
	       "\t-f \tDictionary file\n"
	       "\t-d \tOutput hash file\n"
	       "\t-s \tNetwork SSID\n"
	       "\t-h \tPrint this help information and exit\n"
	       "\t-v \tPrint verbose information (more -v for more verbosity)\n"
	       "\t-V \tPrint program version and exit\n" "\n");
	printf("After precomputing the hash file, run cowpatty with the -d "
		"argument.\n");
}

void cleanup()
{
	/* lame-o-meter++ */
	sig = 1;
}


int nextword(char *word)
{

      strncpy(word,"hellomynameis",13);
      printf("word is: %s",word);
      /*	 
      if (fgets(word, MAXPASSLEN + 1, fp) == NULL) {
		return (-1);
	}

	 //Remove newline 
	word[strlen(word) - 1] = '\0';

	if (feof(fp)) {
		return (-1);
	}
*/
	return (strlen(word));
}

int main(int argc, char **argv)
{
	int fret = 0;
	unsigned long int wordstested=0;
	float elapsed = 0;
	char passphrase[MAXPASSLEN + 1];
	struct user_opt opt;
	struct hashdb_head hf_header;
	//struct hashdb_rec  rec;
	struct stat teststat;
	FILE *fpout = NULL;
	struct timeval start, end;
	//u8 pmk[32];

	printf("%s %s - WPA-PSK precomputation attack. <jwright@hasborg.com>\n",
	       PROGNAME, VER);

	memset(&opt, 0, sizeof(opt));
	memset(&hf_header, 0, sizeof(hf_header));

//	signal(SIGINT, cleanup);
//	signal(SIGTERM, cleanup);
//	signal(SIGQUIT, cleanup);

	/* Collect and test command-line arguments */
	strcpy(opt.hashfile, "sunjayout");
	strcpy(opt.ssid, "ATT256");


	if (IsBlank(opt.hashfile)) {
	   usage("Must specify an output hasfile with -d");
	   exit(1);
	}

	if (IsBlank(opt.ssid)) {
	   usage("Must specify a SSID with -s");
	   exit(1);
	}

	/* stat the hashfile, if it exists, print a message and check to
	   ensure specified SSID matches header information.  If so, append
	   new words to the end of the hashdb file.
	   If the file does not exist, populate the hashdb_head record and
	   create the file. */
	//ret = stat(opt.hashfile, &teststat);
	stat(opt.hashfile,&teststat);
	if (errno == ENOENT || teststat.st_size == 0) {
	   /* File does not exist or is empty, populate header and 
	      create */
	   printf("File %s does not exist, creating.\n", opt.hashfile);
	   memcpy(hf_header.ssid, opt.ssid, strlen(opt.ssid));
	   hf_header.ssidlen = strlen(opt.ssid);
	   hf_header.magic = GENPMKMAGIC;

	   fpout = fopen(opt.hashfile, "wb");
	   if (fpout == NULL) {
	      perror("fopen");
	      exit(-1);
	   }

	   if (fwrite(&hf_header, sizeof(hf_header), 1, fpout) != 1) {
	      perror("fwrite");
	      exit(-1);
	   }

	} else {

	   /* File does exist, append to EOF after matching SSID */
	   fpout = fopen(opt.hashfile, "r+b");
	   if (fpout == NULL) {
	      perror("fopen");
	      exit(-1);
	   }

	   if (fread(&hf_header, sizeof(hf_header), 1, fpout) != 1) {
	      perror("fread");
	      exit(-1);
	   }

	   fclose(fpout);

	   if (memcmp(opt.ssid, hf_header.ssid, hf_header.ssidlen) != 0) {
	      fprintf(stderr, "Specified SSID \"%s\" and the SSID in "
		    "the output file (\"%s\") do not match.\nCreate"
		    " a new file, or change SSID to match.\n",
		    opt.ssid, hf_header.ssid);
	      exit(-1);
	   }

	   printf("File %s exists, appending new data.\n", opt.hashfile);
	   fopen(opt.hashfile, "ab");
	   if (fopen == NULL) {
	      perror("fopen");
	      exit(-1);
	   }
	}



	/* Populate capdata struct */

	gettimeofday(&start, 0);
	int numWords=1;

	pmk *pmks;
	pw *passphrases;
	passphrases = (pw*) calloc(numWords,sizeof(pw));
	pmks = (pmk*) calloc(numWords,sizeof(pmk));

	int i;
	//for(i=0;i<numWords;i++)
	//{
	fret=nextword(passphrases[0].pass);
	//}	   
	if (opt.verbose > 1) {
	   printf("Testing passphrase: %s\n", passphrase);
	}

	/*
	 * Test length of word.  IEEE 802.11i indicates the passphrase must be
	 * at least 8 characters in length, and no more than 63 characters in
	 * length. 
	 */
	if (fret < 8 || fret > 63) {
	   if (opt.verbose) {
	      printf("Invalid passphrase length: %s (%d).\n",
		    passphrase, (int)strlen(passphrase));
	   }
	} else {
	   /* This word is good, increment the words tested counter */
	   wordstested++;
	}

	/* Status display */
	if ((wordstested % 1000) == 0) {
	   printf("key no. %ld: %s\n", wordstested, passphrase);
	   fflush(stdout);
	}

	if (opt.verbose > 1) {
	   printf("Calculating PMK for \"%s\".\n", passphrase);
	}

	printf("launching kernel\n");
	pw *pass_d;
	pmk *pmks_d;
	char *ssid_d;

	//pw *orig_pass = passphrases;
	//pmk *orig_pmks = pmks;


	hipMalloc((void **) &pass_d, sizeof(pw));
	hipMalloc((void **) &pmks_d, sizeof(pmk));
	hipMalloc((void **) &ssid_d, strlen(opt.ssid)+1);


	hipMemcpy(pass_d, passphrases,sizeof(pw), hipMemcpyHostToDevice);
	hipMemcpy(pmks_d, pmks, sizeof(pmk), hipMemcpyHostToDevice);
	hipMemcpy(ssid_d, opt.ssid, strlen(opt.ssid)+1, hipMemcpyHostToDevice);

	pbkdf2_sha1 <<< MAGIC_NUMBER/BLOCKSIZE, BLOCKSIZE >>> (pass_d, ssid_d, strlen(opt.ssid), pmks_d);

	hipMemcpy(pmks, pmks_d,sizeof(pmk), hipMemcpyDeviceToHost);
	int j;
	for (j = 0; j < 32; j++) 
	   printf("%02X", pmks[0].key[j]);
      
//	passphrases += ((numWords>MAGIC_NUMBER)?MAGIC_NUMBER:numWords);
//	pmks += ((numWords>MAGIC_NUMBER)?MAGIC_NUMBER:numWords);
	//numWords-=MAGIC_NUMBER;

//	passphrases = orig_pass;
//	pmks = orig_pmks;

	/*	for (i = 0; i < wordstested; i++) {*/
	printf("record size:%u, password:\"%s\" pmk:", (uint)(strlen(passphrases[0].pass) + sizeof(pmks[0].key) + 1), passphrases[0].pass);
	for (j = 0; j < 32; j++) 
	   printf("%02X", pmks[0].key[j]);
	printf("\n");
	//	}

	//for (i = 0; i < wordstested; i++) {
	/* Write the record contents to the file */
	u8 size = strlen(passphrases[0].pass) + sizeof(pmks[0].key) + 1;
	if (fwrite(&size, 1 , 1, fpout) != 1) {
	   perror("fwrite");
	}
	if (fwrite(passphrases[0].pass, strlen(passphrases[0].pass), 1, fpout) != 1) {
	   perror("fwrite");
	}
	if (fwrite(pmks[0].key, sizeof(pmks[0].key), 1, fpout) != 1) {
	   perror("fwrite");
	}
	//	}
	gettimeofday(&end, 0);

	/* print time elapsed */
	if (end.tv_usec < start.tv_usec) {
	   end.tv_sec -= 1;
	   end.tv_usec += 1000000;
	}
	end.tv_sec -= start.tv_sec;
	end.tv_usec -= start.tv_usec;
	elapsed = end.tv_sec + end.tv_usec / 1000000.0;

	printf("\n%lu passphrases tested in %.2f seconds:  %.2f passphrases/"
	      "second\n", wordstested, elapsed, wordstested / elapsed);

	return (0);
}
