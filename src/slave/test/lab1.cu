#include "hip/hip_runtime.h"
#include "lab1.h"

#ifdef DOUBLE
__global__ void mmKernel(double *Md, double *Nd, double *Pd, int width)
#else
__global__ void mmKernel(float *Md, float *Nd, float *Pd, int width)
#endif
{
   //Calculate the row index of the Pd element and M
   int Row = blockIdx.y * TILE_WIDTH + threadIdx.y;
   // Calculate the column idenx of Pd and N
   int Col = blockIdx.x * TILE_WIDTH + threadIdx.x;

   if (Row * width + Col > width * width)
      return;
   
   #ifdef DOUBLE
   double Pvalue = 0.0;
   #else
   float Pvalue = 0.0;
   #endif

   for (int k = 0; k < width; k++) {
      #ifdef DOUBLE
      double Melem = Md[Row * width + k];
      double Nelem = Nd[k * width + Col];
      #else
      float Melem = Md[Row * width + k];
      float Nelem = Nd[k * width + Col];
      #endif

      Pvalue += Melem * Nelem;
   }
   Pd[Row* width + Col] = Pvalue;
}


matrix_t MMonDevice(matrix_t matrix1, matrix_t matrix2) {
   int blocks;
   blocks=1;
   int width = matrix1.rows;
   int size = width * width;
   matrix_t retMatrix;
   #ifdef DOUBLE
   double *Md, *Nd, *Pd, *P;
   size = size * sizeof(double);
   P = (double *) malloc(size);
   #else
   float *Md, *Nd, *Pd, *P;
   size = size * sizeof(float);
   P = (float *) malloc(size);
   #endif
   dim3 dimBlock;

   hipMalloc(&Md, size);
   hipMemcpy(Md, matrix1.array, size, hipMemcpyHostToDevice);

   hipMalloc(&Nd, size);
   hipMemcpy(Nd, matrix2.array, size, hipMemcpyHostToDevice);

   hipMalloc(&Pd, size);

   blocks = width / TILE_WIDTH;
   //if (width % TILE_WIDTH > 0)
     // blocks++;
   printf("Number of blocks is: %d\n",blocks); 
   //invoke kernel
   // we probably need more than 1 block
   dim3 dimGrid(blocks, blocks);
   
   //width*width must be less than 1024
   if (width < TILE_WIDTH) {
      dimBlock.x = width;
      dimBlock.y = width;
   }
   else {
      dimBlock.x = TILE_WIDTH;
      dimBlock.y = TILE_WIDTH;
   }

   printf("Before mmKernel call\n");
   mmKernel<<<dimGrid, dimBlock>>>(Md, Nd, Pd, width);

   printf("array mmKernel call\n");
   //copy back
   hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost);
   
   hipFree(Md);
   hipFree(Nd);
   hipFree(Pd);

   retMatrix.array = P;
   retMatrix.rows = width;
   retMatrix.cols = width;

   return retMatrix;
}

int main(int argc, char* argv[])
{
   matrix_t matrix1, matrix2, matrix3;

   matrix1 = matrix_read(argv[1]);   
   matrix2 = matrix_read(argv[2]);   

   if(!check_dimensions(matrix1, matrix2)) {
      printf("Matrix dimensions don't match: m1 %d %d m2 %d %d\n", 
            matrix1.rows, matrix1.cols, matrix2.rows, matrix2.cols);
      exit(1);
   }

   matrix3 = MMonDevice(matrix1, matrix2);
   print_matrix(matrix3);

   free(matrix1.array);
   free(matrix2.array);
   free(matrix3.array);
   return 0;
}

matrix_t matrix_read(char* filename)
{
   //pointer to a new line
   matrix_t matrix;
   int columns = 0;
   int rows = 0;

   /* The file descriptor. */
   int fd;
   /* Information about the file. */
   struct stat s;
   int status;
   size_t size;
   /* The memory-mapped thing itself. */
   void* mapped;
   int i;
   int counter;
   counter=0;

   /* Open the file for reading. */
   fd = open (filename, O_RDONLY);

   /* Get the size of the file. */
   fstat (fd, &s);

   /* Memory-map the file. */
   mapped = mmap (0, s.st_size, PROT_READ, MAP_SHARED, fd, 0);
   check (mapped == MAP_FAILED, "mmap %s failed: %s",
	 file_name, strerror (errno));

   /* Now do something with the information. */
   for (i = 0; i < size; i++) {


      if((char*) mapped[i]=='\n')
      {
	 printf("number of numbers in a row is %d\n",counter);
	 break;
      }
      else if((char*) mapped[i]==32)
      {
	 counter++;
      }

   }

   return 0;





/*completely working code
 * *****************************************************************/
   struct stat sb;
   //mmap pointer
   void* file_memory;
   //back up of mmap pointer
   void* backup_file_memory;
   //temp str used for str tok
   char* temp_str;


   /*character used to parse first line*/
   char c;
   char* ptr;
   //file descriptor
   int fd;
   //loop counters
   int j;
   int i;

   fd = open (filename, O_RDONLY);
   // figure out the size
   fstat(fd, &sb);

   file_memory = mmap(0, sb.st_size, PROT_READ, MAP_PRIVATE, fd, 0);
   if(file_memory==MAP_FAILED)
   {

      close (fd);
      fprintf(stderr,"mmap failed. Exitting\n");
      exit(-1);
   }
   close (fd);

   //save original location 
   backup_file_memory=file_memory;

   i=0; 
/*find out how many numbers there are before the newline */ 
   do{
      sscanf ((char*) file_memory, "%c", &c);
      file_memory+=sizeof(char);

      if(c==' ')
      {
	 i++;
      }
      else if(c=='.')
      {
	 //if we are to assume always 2 digits after decimal
	 file_memory=(2*sizeof(char))+file_memory;
      }
      else if(c=='-')
      {
	 //if we assume guaranteed decimal with 2 digits after it
	 file_memory=(4*sizeof(char))+file_memory;
      }

   }while(c!='\n');
   
   
  /*we have to malloc to insert a null character*/
  /*without null we don't know where to stop reading floats or doubles*/ 
  ptr = (char *)malloc( sb.st_size + 1);
   /*copy the memory*/
  memcpy(ptr, backup_file_memory, sb.st_size);
  /*add the null character*/
  ptr[sb.st_size] = '\0';

  /*we no longer need mmapped memory*/
  munmap(file_memory, sb.st_size);
   //munmap(file_memory, sb.st_size);
   
  /*malloc for matrix*/ 
  float* array = (float *)malloc(sizeof(float) * (i*i));
   if (array == NULL) {
      perror("No space to allocate matrix.");
      exit(1);
   }
   /*parse string using tokens*/
   temp_str=strtok(ptr," \n");
   for(i=0;i<fd;i++)
   {
      for(j=0;j<fd;j++)
      {
	 //copy string token into matrix
	 sscanf(temp_str,"%f",&array[i*fd+j]);
	 temp_str=strtok(NULL," \n");
      }
   }
   /*
   for(i=0;i<fd;i++)
   {
      for(j=0;j<fd;j++)
      {
	 printf("%f\n",array[i*fd+j]);
      }
   }
*/

     return 0;
}

/*end completely working code
 * ****************************************************************/





#ifdef DOUBLE
   void *array;
#else
   float *array;
#endif 

   fp = fopen(filename, "r");
   if(fp == NULL)
   {
      fprintf(stderr,"Error opening file\n");
      exit(-1);
   }

   //square matrix
   rows=columns;
   /*
#ifdef DOUBLE
array = (double *) calloc(rows * columns,sizeof(double));
#else
array = (float *) calloc(rows * columns,sizeof(float));
#endif

   //store numbers from matrix
#ifdef DOUBLE
while(fscanf(fp, "%lf", &array[counter])==1) 
#else
while(fscanf(fp, "%f", &array[counter])==1)
#endif
{
counter++;
}

fclose(fp);
    */
   matrix.rows = rows;
   matrix.cols = columns;
   matrix.array = array;

   return matrix;
   }
//return if the two matrices have valid dimensions
int check_dimensions(matrix_t matrix1,matrix_t matrix2)
{

   return matrix1.cols==matrix2.rows;
}
//multiply the matrices together
matrix_t multiply_matrices(matrix_t matrix1, matrix_t matrix2)
{
   //loop counters
   int i;
   int inner;
   int row, col;
   //output matrix
   matrix_t matrix;
   //dot product sum
#ifdef DOUBLE
   double sum;
   matrix.array = (double *) calloc(matrix1.rows * matrix2.cols, sizeof(double));
#else
   float sum;
   matrix.array = (float *) calloc(matrix1.rows * matrix2.cols, sizeof(float));
#endif

   matrix.rows = matrix1.rows;
   matrix.cols = matrix2.cols;
   for(i = 0; i < matrix1.rows * matrix2.cols; i++)
   {
      sum = 0.0;

      //find position in resulting matrix
      col = i % matrix2.cols;
      row = i / matrix2.cols;
      for(inner = 0; inner < matrix1.cols; inner++)
      {
	 sum += matrix1.array[row * matrix1.cols + inner]
	    * matrix2.array[inner * matrix2.cols + col];
      }

      matrix.array[row * matrix.cols + col] = sum;
   }

   return matrix;
}
//print result matrix to 'result.out'
void print_matrix(matrix_t matrix) {
   int i,j;
   FILE *fp;

   fp = fopen("result.out", "w");

   for (i = 0; i < matrix.rows; i++) {
      for (j = 0; j < matrix.cols; j++) {
#ifdef DOUBLE
	 fprintf(fp, "%.2f ", matrix.array[i * matrix.cols + j]);
#else
	 fprintf(fp, "%.2f ", matrix.array[i * matrix.cols + j]);
#endif
      }
      fprintf(fp, "\n");
   }
   fclose(fp);
}
