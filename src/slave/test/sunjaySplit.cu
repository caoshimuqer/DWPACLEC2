matrix_t MMonDevice(matrix_t matrix1, matrix_t matrix2) {
   int blocks;
   blocks=1;
   int width = matrix1.rows;
   int size = width * width;
   matrix_t retMatrix;
   #ifdef DOUBLE
   double *Md, *Nd, *Pd, *P;
   size = size * sizeof(double);
   P = (double *) malloc(size);
   #else
   float *Md, *Nd, *Pd, *P;
   size = size * sizeof(float);
   P = (float *) malloc(size);
   #endif
   dim3 dimBlock;

   hipMalloc(&Md, size);
   hipMemcpy(Md, matrix1.array, size, hipMemcpyHostToDevice);

   hipMalloc(&Nd, size);
   hipMemcpy(Nd, matrix2.array, size, hipMemcpyHostToDevice);

   hipMalloc(&Pd, size);

   blocks = width / TILE_WIDTH;
   //if (width % TILE_WIDTH > 0)
     // blocks++;
   printf("Number of blocks is: %d\n",blocks); 
   //invoke kernel
   // we probably need more than 1 block
   dim3 dimGrid(blocks, blocks);
   
   //width*width must be less than 1024
   if (width < TILE_WIDTH) {
      dimBlock.x = width;
      dimBlock.y = width;
   }
   else {
      dimBlock.x = TILE_WIDTH;
      dimBlock.y = TILE_WIDTH;
   }

   printf("Before mmKernel call\n");
   mmKernel<<<dimGrid, dimBlock>>>(Md, Nd, Pd, width);

   printf("array mmKernel call\n");
   //copy back
   hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost);
   
   hipFree(Md);
   hipFree(Nd);
   hipFree(Pd);

   retMatrix.array = P;
   retMatrix.rows = width;
   retMatrix.cols = width;

   return retMatrix;
}

int main(int argc, char* argv[])
{
   matrix3 = MMonDevice(matrix1, matrix2);
   print_matrix(matrix3);

   free(matrix1.array);
   free(matrix2.array);
   free(matrix3.array);
   return 0;
}
